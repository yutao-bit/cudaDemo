#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "stdio.h"

__device__ float get_element(Matrix* A, int row, int col){
    return A->elements[row * A->width + col];
}
__device__ void set_element(Matrix* C, int row, int col, float value){
    C->elements[row * C->width + col] = value;
}

__global__ void vector_add(Matrix* C, Matrix* A, Matrix* B){
    int indexCol = threadIdx.x + blockIdx.x * blockDim.x;
    int indexRow = threadIdx.y + blockIdx.y * blockDim.y;
    if(indexCol >= C->width || indexRow >= C->height) {
        //printf("%d %d \n", indexCol, indexRow);
        return;
    }

	float cValue = 0.0;
    for(int i = 0; i < A->width; i++){
        cValue += get_element(A, indexRow, i) * get_element(B, i, indexCol);
    }
    set_element(C, indexRow, indexCol, cValue);
}

void vector_add_wrapper(Matrix* C, Matrix* A, Matrix* B){
    if(A->width != B->height || A->height != C->height || B->width != C->width) return ;
    //以输出的矩阵C的宽高元素数量创建线程
    dim3 blockSize(32, 32);
    dim3 girdSize((C->width + blockSize.x - 1) / 32, (C->height + blockSize.y - 1) / 32);
    vector_add<<<girdSize,blockSize>>>(C, A, B);
}